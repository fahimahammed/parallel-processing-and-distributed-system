#include "hip/hip_runtime.h"
% % cu
#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>
#include <sstream>

    using namespace std;

// CUDA kernel for searching a phonebook for a specific name
__global__ void searchPhonebook(const char *data, int totalLength, const char *searchName, int searchNameLength, int *results)
{
    // Thread index and stride calculation for parallel processing
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Iterate through the data in parallel
    for (int i = index; i < totalLength - searchNameLength; i += stride)
    {
        // Check if the current substring matches the search name
        bool found = true;
        for (int j = 0; j < searchNameLength; j++)
        {
            if (data[i + j] != searchName[j])
            {
                found = false;
                break;
            }
        }

        // If a match is found and it is a whole word, increment the results count
        if (found && (i == 0 || data[i - 1] == '\n') && (data[i + searchNameLength] == ' '))
        {
            atomicAdd(results, 1);
        }
    }
}

// Function to read the contents of a file into a string
string readFile(const string &filename)
{
    ifstream file(filename);
    stringstream buffer;
    buffer << file.rdbuf();
    return buffer.str();
}

int main()
{
    // Input file containing phonebook data
    string filename = "/content/sample_data/input.txt";

    // Read the phonebook data and define the search name
    string phonebookData = readFile(filename);
    string searchName = "John";

    // CUDA device memory pointers
    char *dData;
    char *dSearchName;
    int *dResults;
    int results = 0;

    // Compute lengths for data and search name
    int dataLength = phonebookData.length();
    int searchNameLength = searchName.length();

    // Allocate device memory and copy data from host to device
    hipMalloc(&dData, dataLength); // hipError_t hipMalloc(void** devPtr, size_t size);
    hipMalloc(&dSearchName, searchNameLength);
    hipMalloc(&dResults, sizeof(int));
    hipMemcpy(dData, phonebookData.c_str(), dataLength, hipMemcpyHostToDevice); // hipError_t hipMemcpy(void* dst, const void* src, size_t count, hipMemcpyKind kind);
    hipMemcpy(dSearchName, searchName.c_str(), searchNameLength, hipMemcpyHostToDevice);
    hipMemcpy(dResults, &results, sizeof(int), hipMemcpyHostToDevice);

    // Configure CUDA kernel execution parameters
    int blockSize = 256;
    int numBlocks = (dataLength + blockSize - 1) / blockSize;

    // CUDA events for measuring execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch the CUDA kernel for searching the phonebook
    searchPhonebook<<<numBlocks, blockSize>>>(dData, dataLength, dSearchName, searchNameLength, dResults); // kernel_name<<<grid_size, block_size>>>(arg1, arg2, ..., argN);

    // Record the end time and synchronize events
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy the results from device to host and print the results
    hipMemcpy(&results, dResults, sizeof(int), hipMemcpyDeviceToHost);
    cout << "Total Time: " << milliseconds << " ms" << endl;
    cout << "Number of matching contacts: " << results << endl;

    // Free device memory and destroy CUDA events
    hipFree(dData);
    hipFree(dSearchName);
    hipFree(dResults);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
